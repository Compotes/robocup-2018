#include "hip/hip_runtime.h"
#include "image_processing.cuh"

using namespace std;

__device__ __managed__ int h_columns[1032], h_rows[800];

int max_width(int y){
	return y*0.5365897901-2.6025128518+BALL_TOLERANCE;
}

__global__ void column_kernel(const cv::cuda::PtrStepSz<uchar1> src, int* d_columns) {
	if(src(threadIdx.x, blockIdx.x).x == 255){
		atomicAdd(&d_columns[blockIdx.x], 1);
	}
}

__global__ void row_kernel(const cv::cuda::PtrStepSz<uchar1> src, int* d_rows, int start_x) {
	if(src(threadIdx.x, blockIdx.x+start_x).x == 255){
		atomicAdd(&d_rows[threadIdx.x], 1);
	}
}

__global__ void final_kernel(cv::cuda::PtrStepSz<uchar1> src, int* d_columns, int start_x, int start_y) {
	if(src(threadIdx.x+start_y, blockIdx.x+start_x).x == 255){
		atomicAdd(&d_columns[blockIdx.x], 1);
	}
}

__host__ object object_detect(cv::cuda::GpuMat &src, int min_width, int min_height, int max_pixel) {
	object maximum = {-1,-1,-1,-1}, actual_segment;
	vector < object > column_objects;
	vector < object > row_objects;
	vector < object > final_objects;
	//int h_columns[1032], h_rows[max_pixel];
	//int* d_columns, *d_rows;

	//hipMalloc(&d_columns, sizeof(int)*1032);

    hipMemset(h_columns, 0, 1032*sizeof(int));
    column_kernel<<<1032, max_pixel>>>(src, h_columns);
	//hipMemcpy(&h_columns, d_columns, sizeof(int)*1032, hipMemcpyDeviceToHost);
	//hipFree(d_columns);
	hipDeviceSynchronize();

	actual_segment = {0,0,0,max_pixel};
	int hole = 0;

	for (int i = 0; i < 1032; i++) {
		if (h_columns[i] > min_height) { // 20
			if(hole == 0){
				actual_segment.width ++;
			} else if(hole < 35){
				actual_segment.width += (hole+1);
				hole = 0;
			} else {
				hole = 0;
				if(actual_segment.width != 0) column_objects.push_back(actual_segment);
				actual_segment = {i,0,1,max_pixel};
			}
		} else {
			hole++;
		}
	}
	if(actual_segment.width != 0) column_objects.push_back(actual_segment);

	for (int j = 0; j < column_objects.size(); j++) {

		//hipMalloc(&d_rows, sizeof(int)*max_pixel);
		hipMemset(h_rows, 0, sizeof(int)*max_pixel);
		row_kernel<<<column_objects[j].width, max_pixel>>>(src, h_rows, column_objects[j].x);
		//hipMemcpy(&h_rows, d_rows, sizeof(int)*max_pixel, hipMemcpyDeviceToHost);
		//hipFree(d_rows);
		hipDeviceSynchronize();

		actual_segment = {column_objects[j].x, 0, column_objects[j].width, 0};
		hole = 0;

		for (int i = 0; i < max_pixel; i++) {
			if (h_rows[i] > min_width) { // 20
				if(hole == 0){
					actual_segment.height ++;
				} else if(hole < 10){
					actual_segment.height += (hole+1);
					hole = 0;
				} else {
					hole = 0;
					if(actual_segment.height != 0) row_objects.push_back(actual_segment);
					actual_segment = {column_objects[j].x, i, column_objects[j].width, 1};
				}
			} else {
				hole++;
			}
		}
		if(actual_segment.height != 0) row_objects.push_back(actual_segment);
	}

	for (int j = 0; j < row_objects.size(); j++) {

		//hipMalloc(&d_columns, sizeof(int)*row_objects[j].width);
        hipMemset(h_columns, 0, sizeof(int)*row_objects[j].width);
		final_kernel<<<row_objects[j].width, row_objects[j].height>>>(src, h_columns, row_objects[j].x, row_objects[j].y);
		//cout << j << " h" << row_objects[j].height << endl;
		//cout << j << " y" << row_objects[j].y << endl;
		//hipMemcpy(&h_columns, d_columns, sizeof(int)*row_objects[j].width, hipMemcpyDeviceToHost);
		//hipFree(d_columns);
		hipDeviceSynchronize();

		//actual_segment = {row_objects[j].x, row_objects[j].y, 0, row_objects[j].height};
		int hole = 0;
		int test = 0;
		while(h_columns[test] <= min_height) test++;
		actual_segment = {row_objects[j].x+test, row_objects[j].y, 1, row_objects[j].height};
		for (int i = test; i < row_objects[j].width; i++) {
			if (h_columns[i] > min_height) { // 20
				if(hole == 0){
					actual_segment.width ++;
				} else if(hole < 35){
					actual_segment.width += (hole+1);
					hole = 0;
				} else {
					hole = 0;
					if(actual_segment.width != 0) final_objects.push_back(actual_segment);
					actual_segment = {row_objects[j].x+i, row_objects[j].y, 1, row_objects[j].height};
				}
			} else {
				hole++;
			}
		}
		if(actual_segment.width != 0) final_objects.push_back(actual_segment);

	}

	for (int j = 0; j < final_objects.size(); j++) {
		if(final_objects[j].width*final_objects[j].height > maximum.width*maximum.height && final_objects[j].width < max_width(final_objects[j].y + final_objects[j].height/2)){
			maximum = final_objects[j];
		}
	}

	//if (max_width(maximum_row.start_pointer + maximum_row.size/2) < maximum_column.size) {
	//	return {-1,-1,-1,-1};
	//}
	if(maximum.width*maximum.height > 10) {
		return {maximum.x + maximum.width/2, maximum.y + maximum.height/2, maximum.width, maximum.height};
	}
	return {-1,-1,-1,-1};
}
