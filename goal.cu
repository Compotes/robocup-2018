#include "hip/hip_runtime.h"
#include "goal.cuh"

using namespace std;

struct segment {
	int start_pointer;
	int size;
} maximum_column, actual_segment_column, maximum_row, actual_segment_row;

//int max_width(int y){
//	return y*0.5033116626-7.0541699873+BALL_TOLERANCE;
//}

__global__ void goal_kernel(const cv::cuda::PtrStepSz<uchar1> src, int* d_columns, int* d_rows) {
	if(src(threadIdx.x, blockIdx.x).x == 255){
		atomicAdd(&d_columns[blockIdx.x], 1);
		atomicAdd(&d_rows[threadIdx.x], 1);
	}
}

__host__ object goal_detect(cv::cuda::GpuMat &src, int min_width, int min_height, int max_pixel) {
	int h_columns[1032], h_rows[max_pixel];
	int* d_columns, *d_rows;

	hipMalloc(&d_columns, sizeof(int)*1032);
	hipMalloc(&d_rows, sizeof(int)*max_pixel);
	goal_kernel<<<1032, max_pixel>>>(src, d_columns, d_rows);
	hipMemcpy(&h_columns, d_columns, sizeof(int)*1032, hipMemcpyDeviceToHost);
	hipFree(d_columns);
	hipMemcpy(&h_rows, d_rows, sizeof(int)*max_pixel, hipMemcpyDeviceToHost);
	hipFree(d_rows);

	maximum_column = {0,0};
	actual_segment_column = {0,0};
	int hole = 0;

	for (int i = 0; i < 1032; i++) {
		if (h_columns[i] > min_height) { // 20
			if(hole == 0){
				actual_segment_column.size ++;
			} else if(hole < 35){
				actual_segment_column.size += hole+1;
				hole = 0;
			} else {
				hole = 0;
				if(actual_segment_column.size > maximum_column.size) {
					maximum_column = actual_segment_column;
				}
				actual_segment_column = {i,1};
			}
		} else {
			hole++;
		}
	}
	if(actual_segment_column.size > maximum_column.size) {
		maximum_column = actual_segment_column;
	}

	maximum_row = {0,0};
	actual_segment_row = {0,0};
	hole = 0;

	for (int i = 0; i < max_pixel; i++) {
		if (h_rows[i] > min_width) { // 20
			if(hole == 0){
				actual_segment_row.size ++;
			} else if(hole < 35){
				actual_segment_row.size += hole+1;
				hole = 0;
			} else {
				hole = 0;
				if(actual_segment_row.size > maximum_row.size) {
					maximum_row = actual_segment_row;
				}
				actual_segment_row = {i,1};
			}
		} else {
			hole++;
		}
	}
	if(actual_segment_row.size > maximum_row.size) {
		maximum_row = actual_segment_row;
	}

	//if (max_width(maximum_row.start_pointer + maximum_row.size/2) < maximum_column.size) {
	//	return {-1,-1,-1,-1};
	//}

	if(maximum_column.size > min_width) {
		return {maximum_column.start_pointer + maximum_column.size/2, maximum_row.start_pointer + maximum_row.size/2, maximum_column.size, maximum_row.size};
	}
	return {-1,-1,-1,-1};
}

