#include "hip/hip_runtime.h"
#include "goal.cuh"

using namespace std;

 struct segment {
	long long average;
	int size;
	int sum;
} maximum, actual_segment;

__global__ void goal_kernel(const cv::cuda::PtrStepSz<uchar1> src, int* d_columns) {
	if(src(threadIdx.x, blockIdx.x).x == 255){
		atomicAdd(&d_columns[blockIdx.x], 1);
	}
}

__host__ int goal_detect(cv::cuda::GpuMat &src) {
	int h_columns[1032];
	int* d_columns;

	hipMalloc(&d_columns, sizeof(int)*1032);
	goal_kernel<<<1032, 500>>>(src, d_columns);
	hipMemcpy(&h_columns, d_columns, sizeof(int)*1032, hipMemcpyDeviceToHost);
	hipFree(d_columns);

	maximum = {0,0,0};
	actual_segment = {0,0,0};
	int hole = 0;

	for (int i = 0; i < 1032; i++) {
		if (h_columns[i] > 22) { // 20
			if(hole == 0){
				//actual_segment.average += h_columns[i]*i;
				actual_segment.size ++;
				//actual_segment.sum += h_columns[i];
			} else if(hole < 30){
				//actual_segment.average += h_columns[i]*i;
				actual_segment.size += hole+1;
				//actual_segment.sum += h_columns[i];
				hole = 0;
			} else {
				hole = 0;
				if(actual_segment.size > maximum.size) {
					maximum = actual_segment;
				}
				//actual_segment = {h_columns[i]*i,1,h_columns[i]};
				actual_segment = {i,1,0};
			}
			/*ball_visible = 1;
			center_x += h_columns[i]*i;
			sum += h_columns[i];*/
		} else {
			hole++;
		}
	}
	if(actual_segment.size > maximum.size) {
		maximum = actual_segment;
	}

	if(maximum.size > 38) {
		return maximum.average + maximum.size/2;//maximum.average/maximum.sum;
	}
	return -1;
}

